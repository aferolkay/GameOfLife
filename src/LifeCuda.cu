#include "hip/hip_runtime.h"
#include "LifeCuda.cuh"

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

extern "C" __device__ int countNeighbors(uint8_t *grid, int x, int y, int width, int height)
{
    /*
    int count = 0;
    for (int dx = -1; dx <= 1; ++dx)
    {
        for (int dy = -1; dy <= 1; ++dy)
        {
            if (dx == 0 && dy == 0) continue;
            int nx = (x + dx + width) % width;
            int ny = (y + dy + height) % height;
            count += grid[ny * width + nx];
        }
    }
    return count;
    */
   int count = 0;

	count += grid[(x-1) * (width) + (y)];
	count += grid[(x) * (width) + (y-1)];
	count += grid[(x - 1) * (width) + (y-1)];
	count += grid[(x + 1) * (width) + (y)];

	count += grid[(x) * (width) + (y+1)];
	count += grid[(x+1) * (width) + (y+1)];
	count += grid[(x+1) * (width) + (y-1)];
	count += grid[(x - 1) * (width) + (y+1)];

	return count;
}

extern "C" __global__ void updateKernel(uint8_t *currentGrid, uint8_t *nextGrid, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height && x > 0 && y > 0)
    {
        int neighbors = countNeighbors(currentGrid, x, y, width, height);

        if (currentGrid[idx])
        {
            nextGrid[idx] = (neighbors == 2 || neighbors == 3);
        }
        else
        {
            nextGrid[idx] = (neighbors == 3);
        }
    }
}

extern "C" __global__ void dummyKernel(uint8_t *grid, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        int idx = y * width + x;
        grid[idx] = (idx % 5 < 3) ? 1 : 0; // Repeated pattern: 3 ones followed by 2 zeros
    }
}

extern "C" void launchDummyKernel(
    uint8_t* grid,
    int width, int height,
    int gridX, int gridY,
    int blockX, int blockY)
{
  dim3 gridSize (gridX, gridY);
  dim3 blockSize(blockX, blockY);
  dummyKernel<<<gridSize, blockSize>>>(grid, width, height);
  hipDeviceSynchronize();
}

extern "C" void launchUpdateKernel(
    uint8_t* currentGrid,
    uint8_t* nextGrid,
    int width, int height,
    int gridX, int gridY,
    int blockX, int blockY)
{
  dim3 gridSize (gridX, gridY);
  dim3 blockSize(blockX, blockY);
  updateKernel<<<gridSize, blockSize>>>(currentGrid, nextGrid, width, height);
  hipDeviceSynchronize();
}